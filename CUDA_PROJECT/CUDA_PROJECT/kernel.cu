﻿#include "hip/hip_runtime.h"

#include <stdio.h>

// CUDA Add
extern "C" hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b) 
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

extern "C" hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size) 
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

// Cuda Multiply
__global__ void matmul_kernel(int* A, int* B, int* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int value = 0;
        for (int k = 0; k < N; ++k) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

// Function to call the CUDA kernel for matrix multiplication
extern "C" hipError_t matmulWithCuda(int* C, const int* A, const int* B, unsigned int N) {
    int* d_A = nullptr, * d_B = nullptr, * d_C = nullptr;
    size_t size = N * N * sizeof(int);

    // Allocate memory on device
    hipError_t err = hipMalloc((void**)&d_A, size);
    if (err != hipSuccess) return err;

    err = hipMalloc((void**)&d_B, size);
    if (err != hipSuccess) return err;

    err = hipMalloc((void**)&d_C, size);
    if (err != hipSuccess) return err;

    // Copy data from host to device
    err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) return err;

    err = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) return err;

    // Launch the kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (N + 15) / 16); // Divide N by block size

    matmul_kernel<<<numBlocks, threadsPerBlock >>>(d_A, d_B, d_C, N);

    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) return err;

    // Copy the result from device to host
    err = hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) return err;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return hipSuccess;
}
